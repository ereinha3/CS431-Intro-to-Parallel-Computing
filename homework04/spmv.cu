#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

#include "spmv.h"

#define BLOCKDIM 64

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

template <class T>
__global__ void
spmv_kernel_ell(unsigned int* col_ind, T* vals, int m, int n, int nnz, 
                double* x, double* b)
{    
    T thread_sum = 0.0;

    for (unsigned int i = threadIdx.x; i<n; i += BLOCKDIM){
        if ((blockIdx.x * n + i) < (m * n)) { // Check to ensure within bounds
            unsigned int col = col_ind[blockIdx.x * n + i];
            thread_sum += (vals[blockIdx.x * n + i] * x[col]);
        }
    }

    __shared__ T shared_mem[BLOCKDIM];

    shared_mem[threadIdx.x] = thread_sum;

    __syncthreads();


    // parallel reduction
    for (unsigned int i = (BLOCKDIM >> 1); i > 0; i >>= 1)
    {
        if (threadIdx.x < i){
            shared_mem[threadIdx.x] += shared_mem[threadIdx.x + i];
        }
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        b[blockIdx.x] = shared_mem[0];
    }

}



void spmv_gpu_ell(unsigned int* col_ind, double* vals, int m, int n, int n_new, int nnz, 
                  double* x, double* b)
{
    // timers
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    // GPU execution parameters
    unsigned int blocks = m; 
    unsigned int threads = BLOCKDIM; 
    unsigned int shared = threads * sizeof(double);

    dim3 dimGrid(blocks, 1, 1);
    dim3 dimBlock(threads, 1, 1);

    checkCudaErrors(hipEventRecord(start, 0));
    for(unsigned int i = 0; i < MAX_ITER; i++) {
        hipDeviceSynchronize();
        spmv_kernel_ell<double><<<dimGrid, dimBlock, shared>>>(col_ind, vals, 
                                                               m, n, n_new, nnz, x, b);
        printf("Kernel passed %d times\n", i+1);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("Kernel launch error: %s\n", hipGetErrorString(err));
            break;
        }
    } 
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("  Exec time (per itr): %0.8f s\n", (elapsedTime / 1e3 / MAX_ITER));

}




void allocate_ell_gpu(unsigned int* col_ind, double* vals, int m, int n, int n_new,
                      int nnz, double* x, unsigned int** dev_col_ind, 
                      double** dev_vals, double** dev_x, double** dev_b)
{
    // x -> n
    // b -> m
    CUDA_CHECK(hipMalloc(dev_col_ind, m * n_new * sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(dev_vals, m * n_new * sizeof(double)));
    CUDA_CHECK(hipMalloc(dev_x, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(dev_b, m * sizeof(double)));

    CUDA_CHECK(hipMemcpy(*dev_col_ind, col_ind, m * n_new * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(*dev_vals, vals, m * n_new * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(*dev_x, x, n * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(*dev_b, 0, m * sizeof(double)));

    // col_ind back to the host and print
    // unsigned int* host_col_ind = (unsigned int*)malloc(m * n_new * sizeof(unsigned int));
    // CUDA_CHECK(hipMemcpy(host_col_ind, *dev_col_ind, m * n_new * sizeof(unsigned int), hipMemcpyDeviceToHost));

    // for (int i = 0; i < (m * n_new); i++) {
    //     if (i % n_new == 0) {
    //         fprintf(stdout, "\n");
    //     }
    //     fprintf(stdout, "%d ", host_col_ind[i]);
    // }

    // free(host_col_ind);

    // unsigned int* host_vals = (unsigned int*)malloc(m * n_new * sizeof(unsigned int));
    // CUDA_CHECK(hipMemcpy(host_vals, *dev_vals, m * n_new * sizeof(unsigned int), hipMemcpyDeviceToHost));

    // for (int i = 0; i < (m * n_new); i++) {
    //     if (i % n_new == 0) {
    //         fprintf(stdout, "\n");
    //     }
    //     fprintf(stdout, "%d ", host_vals[i]);
    // }

    // free(host_vals);
}

void allocate_csr_gpu(unsigned int* row_ptr, unsigned int* col_ind, 
                      double* vals, int m, int n, int nnz, double* x, 
                      unsigned int** dev_row_ptr, unsigned int** dev_col_ind,
                      double** dev_vals, double** dev_x, double** dev_b)
{

    hipMalloc(dev_row_ptr, (m+1) * sizeof(unsigned int));
    hipMalloc(dev_col_ind, nnz * sizeof(unsigned int));
    hipMalloc(dev_vals, nnz * sizeof(double));
    hipMalloc(dev_x, n * sizeof(double));
    hipMalloc(dev_b, m * sizeof(double));

    hipMemcpy(*dev_row_ptr, row_ptr, (m+1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(*dev_col_ind, col_ind, nnz * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(*dev_vals, vals, nnz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(*dev_x, x, n * sizeof(double), hipMemcpyHostToDevice);
}

void get_result_gpu(double* dev_b, double* b, int m)
{
    // timers
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;


    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipMemcpy(b, dev_b, sizeof(double) * m, 
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("  Pinned Host to Device bandwidth (GB/s): %f\n",
         (m * sizeof(double)) * 1e-6 / elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

template <class T>
void CopyData(
  T* input,
  unsigned int N,
  unsigned int dsize,
  T** d_in)
{
  // timers
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsedTime;

  // Allocate pinned memory on host (for faster HtoD copy)
  T* h_in_pinned = NULL;
  checkCudaErrors(hipHostMalloc( &h_in_pinned, N * dsize));
  assert(h_in_pinned);
  memcpy(h_in_pinned, input, N * dsize);

  // copy data
  checkCudaErrors(hipMalloc( d_in, N * dsize));
  checkCudaErrors(hipEventRecord(start, 0));
  checkCudaErrors(hipMemcpy(*d_in, h_in_pinned,
                             N * dsize, hipMemcpyHostToDevice));
  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
  printf("  Pinned Device to Host bandwidth (GB/s): %f\n",
         (N * dsize) * 1e-6 / elapsedTime);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}


template <class T>
__global__ void
spmv_kernel(unsigned int* row_ptr, unsigned int* col_ind, T* vals, 
            int m, int n, int nnz, double* x, double* b)
{
    int row = blockIdx.x;
    int row_start = row_ptr[row];
    int row_end = row_ptr[row + 1];
    int num_threads = blockDim.x;

    T temp = 0.0;

    for (int idx = row_start + threadIdx.x; idx < row_end; idx += num_threads) {
        unsigned int col = col_ind[idx]; 
        T value = vals[idx];             
        temp += (value * x[col]); 
    }

    __shared__ double shared_mem[BLOCKDIM];

    shared_mem[threadIdx.x] = temp;

    __syncthreads();

    // parallel reduction
    for (int i = (num_threads >> 1); i > 0; i >>= 1) {
        if (threadIdx.x < i) {
            shared_mem[threadIdx.x] += shared_mem[threadIdx.x + i];
        }
    }
    
    __syncthreads();

    // result will all be in first thread
    if (threadIdx.x == 0) {
        b[row] = shared_mem[0];
    }
}



void spmv_gpu(unsigned int* row_ptr, unsigned int* col_ind, double* vals,
              int m, int n, int nnz, double* x, double* b)
{
    // timers
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    // GPU execution parameters
    // 1 thread block per row
    // 64 threads working on the non-zeros on the same row
    unsigned int blocks = m; 
    unsigned int threads = BLOCKDIM; 
    unsigned int shared = threads * sizeof(double);

    dim3 dimGrid(blocks, 1, 1);
    dim3 dimBlock(threads, 1, 1);

    checkCudaErrors(hipEventRecord(start, 0));
    for(unsigned int i = 0; i < MAX_ITER; i++) {
        hipDeviceSynchronize();
        spmv_kernel<double><<<dimGrid, dimBlock, shared>>>(row_ptr, col_ind, 
                                                           vals, m, n, nnz, 
                                                           x, b);
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("  Exec time (per itr): %0.8f s\n", (elapsedTime / 1e3 / MAX_ITER));

}
